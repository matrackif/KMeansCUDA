#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "kmeans.cuh"
#include <io.h>
//#include <sys/types.h>  
//#include <sys/stat.h>
#include <fcntl.h>
//#include <iostream>
#include <string.h> 
#include <time.h>
#define MAX_CHAR_PER_LINE 128

float** readFromFile(int   isBinaryFile,  // flag: 0 or 1 
	char *filename,      // input file name 
	int  *numObjs,       // num of data objects (local) 
	int  *numCoords)     // num of coordinates 
{
	float **objects;
	int     i, j, len;
	size_t numBytesRead;

	if (isBinaryFile) 
	{  // input file is in raw binary format
		int infile;
		if ((infile = open(filename, O_RDONLY, "0600")) == -1) 
		{
			fprintf(stderr, "Error: no such file (%s)\n", filename);
			return NULL;
		}
		numBytesRead = read(infile, numObjs, sizeof(int));
		assert(numBytesRead == sizeof(int));
		numBytesRead = read(infile, numCoords, sizeof(int));
		assert(numBytesRead == sizeof(int));
		
		printf("File %s numObjs   = %d\n", filename, *numObjs);
		printf("File %s numCoords = %d\n", filename, *numCoords);
		

		// allocate space for objects[][] and read all objects 
		len = (*numObjs) * (*numCoords);
		objects = (float**)malloc((*numObjs) * sizeof(float*));
		assert(objects != NULL);
		objects[0] = (float*)malloc(len * sizeof(float));
		assert(objects[0] != NULL);
		//assign the proper pointer to each float* array
		for (i = 1; i < (*numObjs); i++)
		{
			objects[i] = objects[i - 1] + (*numCoords);
		}			
		numBytesRead = read(infile, objects[0], len * sizeof(float));
		assert(numBytesRead == len * sizeof(float));

		close(infile);
	}
	else 
	{  
		// input file is in ASCII format
		FILE *infile;
		char *line, *ret;
		int   lineLen;

		if ((infile = fopen(filename, "r")) == NULL) 
		{
			fprintf(stderr, "Error: no such file (%s)\n", filename);
			return NULL;
		}

		/* first find the number of objects */
		lineLen = MAX_CHAR_PER_LINE;
		line = (char*)malloc(lineLen);
		assert(line != NULL);

		(*numObjs) = 0;
		while (fgets(line, lineLen, infile) != NULL) 
		{
			/* check each line to find the max line length */
			while (strlen(line) == lineLen - 1) 
			{
				/* this line read is not complete */
				len = strlen(line);
				fseek(infile, -len, SEEK_CUR);

				/* increase lineLen */
				lineLen += MAX_CHAR_PER_LINE;
				line = (char*)realloc(line, lineLen);
				assert(line != NULL);

				ret = fgets(line, lineLen, infile);
				assert(ret != NULL);
			}

			if (strtok(line, " \t\n") != 0)
				(*numObjs)++;
		}
		rewind(infile);
		printf("lineLen = %d\n", lineLen);

		// find the num coordinates for each object 
		(*numCoords) = 0;
		while (fgets(line, lineLen, infile) != NULL) 
		{
			if (strtok(line, " \t\n") != 0) 
			{
				/* ignore the id (first coordiinate): numCoords = 1; */
				while (strtok(NULL, " ,\t\n") != NULL) (*numCoords)++;
				break; /* this makes read from 1st object */
			}
		}
		rewind(infile);
		
		printf("File %s numObjs   = %d\n", filename, *numObjs);
		printf("File %s numCoords = %d\n", filename, *numCoords);
		

		// allocate space for objects[][] and read all objects 
		len = (*numObjs) * (*numCoords);
		objects = (float**)malloc((*numObjs) * sizeof(float*));
		assert(objects != NULL);
		objects[0] = (float*)malloc(len * sizeof(float));
		assert(objects[0] != NULL);
		for (i = 1; i < (*numObjs); i++)
		{
			objects[i] = objects[i - 1] + (*numCoords);
		}
			
		i = 0;
		// read all objects 
		while (fgets(line, lineLen, infile) != NULL) 
		{
			if (strtok(line, " \t\n") == NULL) continue;
			for (j = 0; j < (*numCoords); j++)
				objects[i][j] = atof(strtok(NULL, " ,\t\n"));
			i++;
		}

		fclose(infile);
		free(line);
	}

	return objects;
}

// file_write() 
int writeToFile(char      *filename,     // input file name 
	int        numClusters,  // no. clusters 
	int        numObjs,      // no. data objects 
	int        numCoords,    // no. coordinates (local) 
	float    **clusters,     // [numClusters][numCoords] centers 
	int       *membership)   // [numObjs] 
{
	FILE *fptr;
	int   i, j;
	char  outFileName[1024];

	// output: the coordinates of the cluster centres 
	sprintf(outFileName, "%s.cluster_centres", filename);
	printf("Writing coordinates of K=%d cluster centers to file \"%s\"\n",
		numClusters, outFileName);
	fptr = fopen(outFileName, "w");
	for (i = 0; i<numClusters; i++) {
		fprintf(fptr, "%d ", i);
		for (j = 0; j<numCoords; j++)
			fprintf(fptr, "%f ", clusters[i][j]);
		fprintf(fptr, "\n");
	}
	fclose(fptr);

	// output: the closest cluster centre to each of the data points 
	sprintf(outFileName, "%s.membership", filename);
	printf("Writing membership of N=%d data objects to file \"%s\"\n",
		numObjs, outFileName);
	fptr = fopen(outFileName, "w");
	for (i = 0; i<numObjs; i++)
		fprintf(fptr, "%d %d\n", i, membership[i]);
	fclose(fptr);

	return 1;
}
static inline int nextPowerOfTwo(int n) 
{
	n--;

	n = n >> 1 | n;
	n = n >> 2 | n;
	n = n >> 4 | n;
	n = n >> 8 | n;
	n = n >> 16 | n;
	//  n = n >> 32 | n;    //  For 64-bit ints

	return ++n;
}


// square of Euclidian distance between two multi-dimensional points
__host__ __device__ inline static
float eucledianDistanceSquared(int    numCoords,
	int    numObjs,
	int    numClusters,
	float *objects,     // [numCoords][numObjs]
	float *clusters,    // [numCoords][numClusters]
	int    objectId,
	int    clusterId)
{
	int i;
	float ans = 0.0;

	for (i = 0; i < numCoords; i++) 
	{
		//objects objects[numObjs * i + objectId] can be thought of as objects[i][objectId], so the ith coordinate for that objectId
		ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) * (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);		
	}

	return(ans);
}

__global__ static
void findNearestCluster(int numCoords,
	int numObjs,
	int numClusters,
	float *objects,           //  [numCoords][numObjs]
	float *deviceClusters,    //  [numCoords][numClusters]
	int *membership,          //  [numObjs]
	int *intermediates)
{
	extern __shared__ char sharedMemory[];

	//  The type chosen for membershipChanged must be large enough to support
	//  reductions. There are blockDim.x elements, one for each thread in the
	//  block. See numThreadsPerClusterBlock in cuda_kmeans().
	unsigned char *membershipChanged = (unsigned char *)sharedMemory;

	float *clusters = deviceClusters;


	membershipChanged[threadIdx.x] = 0;


	int objectId = blockDim.x * blockIdx.x + threadIdx.x;

	if (objectId < numObjs) 
	{
		int   index, i;
		float dist, min_dist;

		// find the cluster id that has min distance to object 
		index = 0;
		min_dist = eucledianDistanceSquared(numCoords, numObjs, numClusters, objects, clusters, objectId, 0);

		for (i = 1; i < numClusters; i++) 
		{
			dist = eucledianDistanceSquared(numCoords, numObjs, numClusters, objects, clusters, objectId, i);
			// do not need square root 
			if (dist < min_dist) 
			{ // find the min and its array index 
				min_dist = dist;
				index = i;
			}
		}

		if (membership[objectId] != index) 
		{
			membershipChanged[threadIdx.x] = 1;
		}

		/* assign the membership to object objectId */
		membership[objectId] = index;

		__syncthreads();    //  For membershipChanged[]

							//  blockDim.x *must* be a power of two!
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) 
		{
			if (threadIdx.x < s) 
			{
				membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) 
		{
			intermediates[blockIdx.x] = membershipChanged[0];
		}
	}
}
///Performs array reduction on the itermediates
__global__ static
void computeDelta(int *deviceIntermediates,
	int numIntermediates,    //  The actual number of intermediates
	int numIntermediates2)   //  The next power of two
{
	//  The number of elements in this array should be equal to
	//  numIntermediates2, the number of threads launched. It *must* be a power
	//  of two!
	extern __shared__ unsigned int intermediates[];

	//  Copy global intermediate values into shared memory.
	intermediates[threadIdx.x] = (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

	__syncthreads();

	//  numIntermediates2 *must* be a power of two!
	for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) 
	{
		if (threadIdx.x < s) 
		{
			intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) 
	{
		deviceIntermediates[0] = intermediates[0];
	}
}

//  Variable explanation
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//
// return an array of cluster centers of size [numClusters][numCoords]      

float** CudaKmeans(float **objects,      // in: [numObjs][numCoords] 
	int     numCoords,    // num of features for example RGB color is 3 coordinates
	int     numObjs,      // num of objects 
	int     numClusters,  // num of clusters 
	float   threshold,    // objects change membership 
	int    *membership,   // output [numObjs] 
	int    *loop_iterations)
{
	int      i, j, index, loop = 0;
	int     *newClusterSize; // [numClusters]: no. objects assigned in each new cluster 
							 
	float    delta;          // % of objects change their clusters 
	float  **dimObjects;
	float  **clusters;       // out: [numClusters][numCoords] 
	float  **dimClusters;
	float  **newClusters;    // [numCoords][numClusters] 

	float *deviceObjects;
	float *deviceClusters;
	int *deviceMembership;
	int *deviceIntermediates;

	//  Copy objects given in [numObjs][numCoords] layout to new
	//  [numCoords][numObjs] layout
	malloc2D(dimObjects, numCoords, numObjs, float);
	for (i = 0; i < numCoords; i++) 
	{
		for (j = 0; j < numObjs; j++) 
		{
			dimObjects[i][j] = objects[j][i];
		}
	}

	// pick first numClusters elements of objects[] as initial cluster centers
	malloc2D(dimClusters, numCoords, numClusters, float);
	for (i = 0; i < numCoords; i++) 
	{
		for (j = 0; j < numClusters; j++) 
		{
			dimClusters[i][j] = dimObjects[i][j];
		}
	}

	// initialize membership[] 
	for (i = 0; i < numObjs; i++) 
	{
		membership[i] = -1;
	}

	// need to initialize newClusterSize and newClusters[0] to all 0 
	newClusterSize = (int*)calloc(numClusters, sizeof(int));
	assert(newClusterSize != NULL);

	malloc2D(newClusters, numCoords, numClusters, float);
	memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

	//  To support reduction, numThreadsPerClusterBlock *must* be a power of
	//  two, and it must be no larger than the number of bits that will
	//  fit into an unsigned char, the type used to keep track of membership
	//  changes in the kernel.
	const unsigned int numThreadsPerClusterBlock = 128;
	const unsigned int numClusterBlocks = (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
		

	const unsigned int clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(unsigned char);
		
	const unsigned int numReductionThreads = nextPowerOfTwo(numClusterBlocks);
		
	const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);
	

	checkCuda(hipMalloc(&deviceObjects, numObjs * numCoords * sizeof(float)), "allocating device objects");
	checkCuda(hipMalloc(&deviceClusters, numClusters * numCoords * sizeof(float)), "allocating device clusters");
	checkCuda(hipMalloc(&deviceMembership, numObjs * sizeof(int)), "allocating device memship");
	checkCuda(hipMalloc(&deviceIntermediates, numReductionThreads * sizeof(unsigned int)), "allocating device intermediates");

	checkCuda(hipMemcpy(deviceObjects, dimObjects[0],	numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice), "memcpy dimObjects[0] -> device objects");
	checkCuda(hipMemcpy(deviceMembership, membership,	numObjs*sizeof(int), hipMemcpyHostToDevice), "memcpy membership -> device membership");

	do 
	{
		checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice), "memcpy dimClusters[0] -> device clusters");
		printf("(for findNearestCluster) numClusterBlocks: %d, numThreadsPerClusterBlock: %d, clusterBlockSharedDataSize: %d \n", numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize);
		findNearestCluster
			<< < numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >> >(numCoords, numObjs, numClusters, deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

		hipDeviceSynchronize(); 
		checkLastCudaError("findNearestCluster");
		printf("(for computeDelta) numReductionThreads: %d, reductionBlockSharedDataSize: %d \n", numReductionThreads, reductionBlockSharedDataSize);
		const int numOfThreadsPerBlock = (numReductionThreads > 1024 ? 1024 : numReductionThreads); // maximum amount of threads per block is 1024
		computeDelta << < 1, numOfThreadsPerBlock, reductionBlockSharedDataSize >> >(deviceIntermediates, numClusterBlocks, numReductionThreads);
		hipDeviceSynchronize(); 
		checkLastCudaError("computeDelta");

		int d;
		checkCuda(hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost), "memcpy deviceIntermediates -> &d");
		delta = (float)d;

		checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost), "memcpy deviceMembership -> membership");

		for (i = 0; i < numObjs; i++) 
		{
			// find the array index of nestest cluster center 
			index = membership[i];

			// update new cluster centers : sum of objects located within
			newClusterSize[index]++;
			for (j = 0; j < numCoords; j++)
				newClusters[j][index] += objects[i][j];
		}

		// average the sum and replace old cluster centers with newClusters 
		for (i = 0; i < numClusters; i++) 
		{
			for (j = 0; j < numCoords; j++) 
			{
				if (newClusterSize[i] > 0) 
				{
					dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
				}					
				newClusters[j][i] = 0.0;   // set back to 0 
			}
			newClusterSize[i] = 0;   // set back to 0 
		}

		delta /= numObjs;
	} while (delta > threshold && loop++ < 500); // 500 is max number of iterations

	*loop_iterations = loop + 1;

	// allocate a 2D space for returning variable clusters[] (coordinatesof cluster centers) 
	
	malloc2D(clusters, numClusters, numCoords, float);
	for (i = 0; i < numClusters; i++) 
	{
		for (j = 0; j < numCoords; j++) 
		{
			clusters[i][j] = dimClusters[j][i];
		}
	}

	checkCuda(hipFree(deviceObjects), "free deviceObjects");
	checkCuda(hipFree(deviceClusters), "free deviceClusters");
	checkCuda(hipFree(deviceMembership), "free deviceMembership");
	checkCuda(hipFree(deviceIntermediates), "free deviceIntermediates");

	free(dimObjects[0]);
	free(dimObjects);
	free(dimClusters[0]);
	free(dimClusters);
	free(newClusters[0]);
	free(newClusters);
	free(newClusterSize);

	return clusters;
}
int main(int argc, char **argv)
{
	int     isBinaryFile, is_output_timing;

	int     numClusters, numCoords, numObjs;
	int    *membership;    // [numObjs] 
	char   *filename;
	float **objects;       // [numObjs][numCoords] data objects 
	float **clusters;      // [numClusters][numCoords] cluster center 
	float   threshold;
	double  timing, io_timing, clustering_timing;
	int     loop_iterations;
	threshold = 0.001;
	numClusters = 0;
	isBinaryFile = 0;
	is_output_timing = 1;
	
	if (argc < 3)
	{
		printf("argv[1] is filename and argv[2] is numOfClusters");
		return -1;
	}
	filename = argv[1];
	numClusters = atoi(argv[2]);
	if (is_output_timing) io_timing = clock();

	// read data points from file 
	objects = readFromFile(isBinaryFile, filename, &numObjs, &numCoords);
	if (objects == NULL) exit(1);

	if (is_output_timing) 
	{
		timing = clock();
		io_timing = timing - io_timing;
		clustering_timing = timing;
	}

	// start the timer for the core computation
	// membership: the cluster id for each data object 
	membership = (int*)malloc(numObjs * sizeof(int));
	assert(membership != NULL);

	clusters = CudaKmeans(objects, numCoords, numObjs, numClusters, threshold,
		membership, &loop_iterations);

	free(objects[0]);
	free(objects);

	if (is_output_timing) 
	{
		timing = clock();
		clustering_timing = (timing - clustering_timing) / CLOCKS_PER_SEC;
	}

	// output: the coordinates of the cluster centres
	writeToFile(filename, numClusters, numObjs, numCoords, clusters,
		membership);

	free(membership);
	free(clusters[0]);
	free(clusters);

	// output performance number..You can also see kernel execution time in Visual Studio 2017
	if (is_output_timing) 
	{
		io_timing += clock() - timing;
		printf("\nPerforming **** Regular Kmeans (CUDA version) ****\n");

		printf("Input file:     %s\n", filename);
		printf("numObjs       = %d\n", numObjs);
		printf("numCoords     = %d\n", numCoords);
		printf("numClusters   = %d\n", numClusters);
		printf("threshold     = %.4f\n", threshold);

		printf("Loop iterations    = %d\n", loop_iterations);

		printf("I/O time           = %10.4f sec\n", io_timing);
		printf("Computation timing = %10.4f sec\n", clustering_timing);
	}

    return 0;
}

